#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <chrono>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
// #include <chrono>
#ifdef GUI
#include <GL/glut.h>
#include <GL/gl.h>
#include <GL/glu.h>
#endif

#include "./headers/physics.h"

int size; // problem size
int block_size = 512; // cuda thread block size

__global__ void initialize(float *data, int size) {
    // TODO: intialize the temperature distribution 
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int n = size * size;
    if (i < n) {
        data[i] = wall_temp;
    }
}

__global__ void generate_fire_area(bool *fire_area, int size) {

    int index = blockDim.x * blockIdx.x + threadIdx.x;
    int n = size * size;
    int x = index/size;
    int y = index%size;

    if (index < n) {
        float fire1_r2 = fire_size * fire_size;
        fire_area[index] = 0;
        int a = x - size / 2;
        int b = y - size / 2;
        int r2 = 0.5 * a * a + 0.8 * b * b - 0.5 * a * b;
        if (r2 < fire1_r2) fire_area[x * size + y] = 1;

        float fire2_r2 = (fire_size / 2) * (fire_size / 2);
        a = x - 1 * size / 3;
        b = y - 1 * size / 3;
        r2 = a * a + b * b;
        if (r2 < fire2_r2) fire_area[x * size + y] = 1;
    }
}

__global__ void update(float *data, float *new_data, int size) {
    
    // TODO: update temperature for each point  
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int n = size * size;

    if (i < n) {
        float up = data[i - size];
        float down = data[i + size];
        float left = data[i - 1];
        float right = data[i + 1];
        float new_val = (up + down + left + right) / 4;
        new_data[i] = new_val;
    }
}

__global__ void maintain_wall(float *data, int size) {
    // TODO: maintain the temperature of the wall 
    for (int i = 0; i < size; i++) {
        data[i] = wall_temp;
        data[i * size] = wall_temp;
        data[i * size + size - 1] = wall_temp;
        data[size * size - 1 - i] = wall_temp;
    }
}

__global__ void maintain_fire(float *data, bool *fire_area,int size) {
    
    // TODO: maintain the temperature of the fire 
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    int length = size * size;
    if (index < length) {  
        if (fire_area[index]) data[index] = fire_temp;
    }
}

#ifdef GUI
__global__ void data2pixels(float *data, GLubyte* pixels){
    // TODO: convert rawdata (large, size^2) to pixels (small, resolution^2) for faster rendering speed (in parallelized way)
}
void plot(GLubyte* pixels) {
    // visualize temprature distribution
    #ifdef GUI
    glClear(GL_COLOR_BUFFER_BIT);
    glDrawPixels(resolution, resolution, GL_RGB, GL_UNSIGNED_BYTE, pixels);
    glutSwapBuffers();
    #endif
}
#endif

void master() {
    
    float *data_odd;
    float *data_even;
    bool *fire_area;

    hipMalloc(&data_odd, size * size * sizeof(float));
    hipMalloc(&data_even, size * size * sizeof(float));
    hipMalloc(&fire_area, size * size * sizeof(bool));

    #ifdef GUI
    GLubyte *pixels;
    GLubyte *host_pixels;
    host_pixels = new GLubyte[resolution * resolution * 3];
    hipMalloc(&pixels, resolution * resolution * 3 * sizeof(GLubyte));
    #endif

    int n_block_size = size * size / block_size + 1;
    int n_block_resolution = resolution * resolution / block_size + 1;

    initialize<<<n_block_size, block_size>>>(data_odd,size);
    generate_fire_area<<<n_block_size, block_size>>>(fire_area,size);

    int count = 1;
    double total_time = 0;

    while (count <= max_iter) {
        std::chrono::high_resolution_clock::time_point t1 = std::chrono::high_resolution_clock::now();
        // TODO: modify the following lines to fit your need.
        if (count % 2 == 1) {
            update<<<n_block_size, block_size>>>(data_odd, data_even,size);
            maintain_fire<<<n_block_size, block_size>>>(data_even, fire_area,size);
            maintain_wall<<<1, 1>>>(data_even,size);
        } else {
            update<<<n_block_size, block_size>>>(data_even, data_odd,size);
            maintain_fire<<<n_block_size, block_size>>>(data_odd, fire_area,size);
            maintain_wall<<<1, 1>>>(data_odd,size);
        }

        std::chrono::high_resolution_clock::time_point t2 = std::chrono::high_resolution_clock::now();
        double this_time = std::chrono::duration<double>(t2 - t1).count();
        total_time += this_time;
        // printf("Iteration %d, elapsed time: %.6f\n",count,this_time);
        
        count++;
        
        #ifdef GUI
        if (count % 2 == 1) {
            data2pixels<<<n_block_resolution, block_size>>>(data_even, pixels);
        } else {
            data2pixels<<<n_block_resolution, block_size>>>(data_odd, pixels);
        }
        hipMemcpy(host_pixels, pixels, resolution * resolution * 3 * sizeof(GLubyte), hipMemcpyDeviceToHost);
        plot(host_pixels);
        #endif
    }

    printf("Converge after %d iterations, elapsed time: %.6f, average computation time: %.6f\n", count-1, total_time, (double) total_time / (count-1));
    
    hipFree(data_odd);
    hipFree(data_even);
    hipFree(fire_area);
    
    #ifdef GUI
    hipFree(pixels);
    delete[] host_pixels;
    #endif   
}


int main(int argc, char *argv[]){
    size = atoi(argv[1]);
    
    #ifdef GUI
    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_RGB | GLUT_SINGLE);
    glutInitWindowPosition(0, 0);
    glutInitWindowSize(resolution, resolution);
    glutCreateWindow("Heat Distribution Simulation CUDA Implementation");
    gluOrtho2D(0, resolution, 0, resolution);
    #endif
    
    master();
    
    printf("Student ID: 119010221\n"); // replace it with your student id
    printf("Name: Haoyan Luo\n"); // replace it with your name
    printf("Assignment 4: Heat Distribution CUDA Implementation\n");
   
    return 0;
}


