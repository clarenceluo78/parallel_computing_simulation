#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <chrono>
#ifdef GUI
#include <GL/glut.h>
#include <GL/gl.h>
#include <GL/glu.h>
#endif

#include "./headers/physics.h"
#include "./headers/logger.h"


int block_size = 512;

std::chrono::high_resolution_clock::time_point start;
std::chrono::high_resolution_clock::time_point end;
int n_body;
int n_iteration;


__global__ void update_position(double *x, double *y, double *vx, double *vy, int n) {

    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) {
        x[i] = x[i] + vx[i]*dt;
        y[i] = y[i] + vy[i]*dt;

        // bound collision check
        if (x[i] <= sqrt(radius2) || x[i] >= bound_x - sqrt(radius2)) vx[i] = -vx[i];
        if (y[i] <= sqrt(radius2) || y[i] >= bound_y - sqrt(radius2)) vy[i] = -vy[i];
    }
}

__global__ void update_velocity(double *m, double *x, double *y, double *vx, double *vy, int n) {
    
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < n) {  
        double deltaX, deltaY, distance, acceleration;

        for (int j = 0; j < n; j++) {
            
            deltaX = x[j] - x[i];
            deltaY = y[j] - y[i];
            distance = sqrt((deltaX * deltaX) + (deltaY * deltaY));

            if (distance < 2 * sqrt(radius2)) {
                vx[i] = -vx[i];
                vy[i] = -vy[i];
                continue;
            }

            acceleration = gravity_const * m[j] / (distance * distance);
            vx[i] += acceleration * deltaX / distance * dt;
            vy[i] += acceleration * deltaY / distance * dt;
            
        }

    }
}


void generate_data(double *m, double *x,double *y,double *vx,double *vy, int n) {
    // TODO: Generate proper initial position and mass for better visualization
    srand((unsigned)time(NULL));
    for (int i = 0; i < n; i++) {
        m[i] = rand() % max_mass + 1.0f;
        x[i] = 1500.0f + rand() % (bound_x / 4);
        y[i] = 1500.0f + rand() % (bound_y / 4);
        vx[i] = 0.0f;
        vy[i] = 0.0f;
    }
}



void master() {
    double* m = new double[n_body];
    double* x = new double[n_body];
    double* y = new double[n_body];
    double* vx = new double[n_body];
    double* vy = new double[n_body];

    generate_data(m, x, y, vx, vy, n_body);

    // Logger l = Logger("cuda", n_body, bound_x, bound_y);

    double *device_m;
    double *device_x;
    double *device_y;
    double *device_vx;
    double *device_vy;

    hipMalloc(&device_m, n_body * sizeof(double));
    hipMalloc(&device_x, n_body * sizeof(double));
    hipMalloc(&device_y, n_body * sizeof(double));
    hipMalloc(&device_vx, n_body * sizeof(double));
    hipMalloc(&device_vy, n_body * sizeof(double));

    hipMemcpy(device_m, m, n_body * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(device_x, x, n_body * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(device_y, y, n_body * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(device_vx, vx, n_body * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(device_vy, vy, n_body * sizeof(double), hipMemcpyHostToDevice);

    int n_block = n_body / block_size + 1;

    for (int i = 0; i < n_iteration; i++){
        // std::chrono::high_resolution_clock::time_point t1 = std::chrono::high_resolution_clock::now();

        update_velocity<<<n_block, block_size>>>(device_m, device_x, device_y, device_vx, device_vy, n_body);
        update_position<<<n_block, block_size>>>(device_x, device_y, device_vx, device_vy, n_body);

        hipMemcpy(x, device_x, n_body * sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(y, device_y, n_body * sizeof(double), hipMemcpyDeviceToHost);

        // std::chrono::high_resolution_clock::time_point t2 = std::chrono::high_resolution_clock::now();
        // std::chrono::duration<double> time_span = t2 - t1;
        
        // printf("Iteration %d, elapsed time: %.3f\n", i, time_span);

        // l.save_frame(x, y);

        #ifdef GUI
        glClear(GL_COLOR_BUFFER_BIT);
        glColor3f(1.0f, 0.0f, 0.0f);
        glPointSize(2.0f);
        glBegin(GL_POINTS);
        double xi;
        double yi;
        for (int i = 0; i < n_body; i++){
            xi = x[i];
            yi = y[i];
            glVertex2f(xi, yi);
        }
        glEnd();
        glFlush();
        glutSwapBuffers();
        #else

        #endif

    }

    hipFree(device_m);
    hipFree(device_x);
    hipFree(device_y);
    hipFree(device_vx);
    hipFree(device_vy);

    delete[] m;
    delete[] x;
    delete[] y;
    delete[] vx;
    delete[] vy;
    
}


int main(int argc, char *argv[]){
    
    n_body = atoi(argv[1]);
    n_iteration = atoi(argv[2]);

    #ifdef GUI
    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_RGB | GLUT_SINGLE);
    glutInitWindowPosition(0, 0);
    glutInitWindowSize(500, 500);
    glutCreateWindow("N Body Simulation CUDA Implementation");
    glClearColor(0.0f, 0.0f, 0.0f, 1.0f);
    gluOrtho2D(0, bound_x, 0, bound_y);
    #endif
    start = std::chrono::high_resolution_clock::now();
    master();
    end = std::chrono::high_resolution_clock::now();
    printf("Student ID: 119010221\n"); // replace it with your student id
    printf("Name: Haoyan Luo\n"); // replace it with your name
    printf("Assignment 3: N Body Simulation CUDA Implementation\n");
    std::chrono::duration<double> total_time = end - start;
    printf("Total time: %.3f\n", total_time);
    return 0;

}


